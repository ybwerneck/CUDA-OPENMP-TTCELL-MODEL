#include "hip/hip_runtime.h"
﻿// cardiac-cell-solver.cpp : Este arquivo contém a função 'main'. A execução do programa começa e termina ali.
//


#include <string>
#ifdef  OSisWindows
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.7\include\hip/hip_runtime.h" // RASCUNHO
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.7\include\"
#else
#include "hip/hip_runtime.h"
#include ""

#endif
#include <vector>
#include <map>
#include <list>
#include <iostream>
#include <fstream>
#include <sstream>

#include "defs.h"






namespace CommandLineProcessing
{

	class ArgvParser
	{
	public:
		typedef int OptionAttributes;
		typedef int ParserResults;
		typedef std::map<std::string, unsigned int> String2KeyMap;
		typedef std::map<unsigned int, OptionAttributes> Key2AttributeMap;
		typedef std::map<unsigned int, std::string> Key2StringMap;
		typedef std::vector<std::string> ArgumentContainer;

		ArgvParser();
		~ArgvParser();

		/** Attributes for options. */
		enum
		{
			NoOptionAttribute = 0x00,
			OptionRequiresValue = 0x01,
			OptionRequired = 0x02
		};
		/** Return values of the parser. */
		enum
		{
			NoParserError = 0x00,
			ParserUnknownOption = 0x01,
			ParserMissingValue = 0x02,
			ParserOptionAfterArgument = 0x04,
			ParserMalformedMultipleShortOption = 0x08,
			ParserRequiredOptionMissing = 0x16,
			ParserHelpRequested = 0x32
		};

		/** Defines an option with optional attributes (required, ...) and an
		* additional (also optional) description. The description becomes part of the
		* generated usage help that can be requested by calling the usageDescription()
		* method.
		* \return Returns FALSE if there already is an option with this name
		* OR if a short option string (length == 1) is a digit. In that case no
		* action is peformed.
		*/
		bool defineOption(const std::string& _name,
			const std::string& _description = std::string(),
			OptionAttributes _attributes = NoOptionAttribute);
		/** Define an alternative name for an option that was previously defined by
		* defineOption().
		* \return Returns FALSE if there already is an option with the alternative
		* name or no option with the original name OR if a short option string
		* (length == 1) is a digit. In that case no action is performed.
		*/
		bool defineOptionAlternative(const std::string& _original,
			const std::string& _alternative);
		/** Returns whether _name is a defined option. */
		bool isDefinedOption(const std::string& _name) const;
		/** Returns whether _name is an option that was found while parsing
		* the command line arguments with the parse() method. In other word: This
		* method returns true if the string is an option AND it was given on the
		* parsed command line.
		*/
		bool foundOption(const std::string& _name) const;
		/** Define a help option. If this option is found a special error code is
		* returned by the parse method.
		* \attention If this method is called twice without an intermediate call
		* to the reset() method the previously set help option will remain a valid
		* option but is not detected as the special help option and will therefore
		* not cause the parse() method to return the special help error code.
		* \return Returns FALSE if there already is an option defined that equals
		* the short or long name.
		*/
		bool setHelpOption(const std::string& _longname = "h",
			const std::string& _shortname = "help",
			const std::string& _descr = "");
		/** Returns the number of read arguments. Arguments are efined as beeing
		* neither options nor option values and are specified at the end of the
		* command line after all options and their values. */
		unsigned int arguments() const;
		/** Returns the Nth argument. See arguments().
		* \return Argument string or an empty string if there was no argument of
		* that id.
		*/
		std::string argument(unsigned int _number) const;
		/** Get the complete argument vector. The order of the arguments in the
		* vector is the same as on the commandline.
		*/
		const std::vector<std::string>& allArguments() const;
		/** Add an error code and its description to the command line parser.
		* This will do nothing more than adding an entry to the usage description.
		*/
		void addErrorCode(int _code, const std::string& _descr = "");
		/** Set some string as a general description, that will be printed before
		* the list of available options.
		*/
		void setIntroductoryDescription(const std::string& _descr);
		/** Parse the command line arguments for all known options and arguments.
		* \return Error code with parsing result.
		* \retval NoParserError Everything went fine.
		* \retval ParserUnknownOption Unknown option was found.
		* \retval ParserMissingValue A value to a given option is missing.
		* \retval ParserOptionAfterArgument Option after an argument detected. All
		* options have to given before the first argument.
		* \retval ParserMalformedMultipleShortOption Malformed short option string.
		* \retval ParserRequiredOptionMissing Required option is missing.
		* \retval ParserHelpRequested Help option detected.
		*/
		ParserResults parse(int _argc, char** _argv);
		/** Return the value of an option.
		* \return Value of a commandline options given by the name of the option or
		* an empty string if there was no such option or the option required no
		* value.
		*/
		std::string optionValue(const std::string& _option) const;
		/** Reset the parser. Call this function if you want to parse another set of
		* command line arguments with the same parser object.
		*/
		void reset();
		/** Returns the name of the option that was responsible for a parser error.
		  * An empty string is returned if no error occured at all.
		  */
		const std::string& errorOption() const;
		/** This method can be used to evaluate parser error codes and generate a
		* human-readable description. In case of a help request error code the
		* usage description as returned by usageDescription() is printed.
		*/
		std::string parseErrorDescription(ParserResults _error_code) const;
		/** Returns a string with the usage descriptions for all options. The
		 * description string is formated to fit into a terminal of width _width.*/

		 /** Returns the key of a defined option with name _name or -1 if such option
		  * is not defined. */
		int optionKey(const std::string& _name) const;
	private:
		/** Returns a list of option names that are all alternative names associated
		 * with a single key value.
		 */
		std::list<std::string> getAllOptionAlternatives(unsigned int _key) const;

		/** The current maximum key value for an option. */
		unsigned int max_key;
		/** Map option names to a numeric key. */
		String2KeyMap option2key;

		/** Map option key to option attributes. */
		Key2AttributeMap option2attribute;

		/** Map option key to option description. */
		Key2StringMap option2descr;

		/** Map option key to option value. */
		Key2StringMap option2value;

		/** Map error code to its description. */
		std::map<int, std::string> errorcode2descr;

		/** Vector of command line arguments. */
		ArgumentContainer argument_container;

		/** General description to be returned as first part of the generated help page. */
		std::string intro_description;

		/** Holds the key for the help option. */
		unsigned int help_option;

		/** Holds the name of the option that was responsible for a parser error.
		*/
		std::string error_option;
	}; // class ArgvParser


	// Auxillary functions

	/** Returns whether the given string is a valid (correct syntax) option string.
	 * It has to fullfill the following criteria:
	 *  1. minimum length is 2 characters
	 *  2. Start with '-'
	 *  3. if if minimal length -> must not be '--'
	 *  4. first short option character must not be a digit (to distinguish negative numbers)
	 */
	bool isValidOptionString(const std::string& _string);

	/** Returns whether the given string is a valid (correct syntax) long option string.
	 * It has to fullfill the following criteria:
	 *  1. minimum length is 4 characters
	 *  2. Start with '--'
	 */
	bool isValidLongOptionString(const std::string& _string);

	/** Splits option and value string if they are given in the form 'option=value'.
	* \return Returns TRUE if a value was found.
	*/
	bool splitOptionAndValue(const std::string& _string, std::string& _option,
		std::string& _value);

	/** String tokenizer using standard C++ functions. Taken from here:
	 * http://gcc.gnu.org/onlinedocs/libstdc++/21_strings/howto.html#3
	 * Splits the string _in by _delimiters and store the tokens in _container.
	 */
	template <typename Container>
	void splitString(Container& _container, const std::string& _in,
		const char* const _delimiters = " \t\n")
	{
		const std::string::size_type len = _in.length();
		std::string::size_type i = 0;

		while (i < len)
		{
			// eat leading whitespace
			i = _in.find_first_not_of(_delimiters, i);
			if (i == std::string::npos)
				return;   // nothing left but white space

			// find the end of the token
			std::string::size_type j = _in.find_first_of(_delimiters, i);

			// push token
			if (j == std::string::npos)
			{
				_container.push_back(_in.substr(i));
				return;
			}
			else
				_container.push_back(_in.substr(i, j - i));

			// set up for next loop
			i = j + 1;
		}
	}

	/** Returns true if the character is a digit (what else?). */
	bool isDigit(const char& _char);

	/** Build a vector of integers from a string of the form:
	* '1,3-5,14,25-20'. This string will be expanded to a list of positive
	* integers with the following elements: 1,3,4,5,14,25,24,23,22,21,20.
	* All of the expanded elements will be added to the provided list.
	* \return Returns FALSE if there was any syntax error in the given string
	* In that case the function stops at the point where the error occured.
	* Only elements processed up to that point will be added to the expanded
	* list.
	* \attention This function can only handle unsigned integers!
	*/
	bool expandRangeStringToUInt(const std::string& _string,
		std::vector<unsigned int>& _expanded);
	/** Returns a copy of _str with whitespace removed from front and back. */
	std::string trimmedString(const std::string& _str);

	/** Formats a string of an arbitrary length to fit a terminal of width
	* _width and to be indented by _indent columns.
	*/
	std::string formatString(const std::string& _string,
		unsigned int _width,
		unsigned int _indent = 0);

};


using namespace std;

using namespace CommandLineProcessing;

class OptionParser {
public:
	OptionParser();

	static void setup();
	static void addOption(const string& option, const string& desc);
	static void parseOptions(int argc, char** argv);
	static string optionValue(const string& option);
	static bool foundOption(const string& option);

	void add(const string& option, const string& desc);
	int parse(string optionsText);
	string value(const string& option);
	bool has(const string& option);

	static float parsefloat(const string& option);
	static int parseInt(const string& option);
private:
	static ArgvParser cmd;
	ArgvParser myCmd;
};


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}	


__device__ float* out_g;
//GPU
__device__ void  partitionedStep(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut = NULL);
__device__ void step(float* Y_new_, int n, float* as, float* bs, float* Y_old_, float dt);
__device__ void step(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut);
__device__ void calc_algs_hh(float* algs, float* pars, float* Y_old_, float time)
{
	calc_xr1_inf = (1.0e+00 / (1.0e+00 + exp((((-2.60e+01) - V_old_) / 7.0e+00))));	//11+
	calc_alpha_xr1 = (4.50e+02 / (1.0e+00 + exp((((-4.50e+01) - V_old_) / 1.0e+01))));	//12
	calc_beta_xr1 = (6.0e+00 / (1.0e+00 + exp(((V_old_ + 3.0e+01) / 1.150e+01))));	//13
	calc_xr2_inf = (1.0e+00 / (1.0e+00 + exp(((V_old_ + 8.80e+01) / 2.40e+01))));	//16
	calc_alpha_xr2 = (3.0e+00 / (1.0e+00 + exp((((-6.0e+01) - V_old_) / 2.0e+01))));	//17
	calc_beta_xr2 = (1.120e+00 / (1.0e+00 + exp(((V_old_ - 6.0e+01) / 2.0e+01))));	//18
	calc_xs_inf = (1.0e+00 / (1.0e+00 + exp((((-5.0e+00) - V_old_) / 1.40e+01))));	//22
	calc_alpha_xs = (1.10e+03 / pow((1.0e+00 + exp((((-1.0e+01) - V_old_) / 6.0e+00))), 1.0 / 2.0));	//23
	calc_beta_xs = (1.0e+00 / (1.0e+00 + exp(((V_old_ - 6.0e+01) / 2.0e+01))));	//24

	calc_m_inf = (1.0e+00 / pow((1.0e+00 + exp((((-5.6860e+01) - V_old_) / 9.03e+00))), 2.0e+00));	//28
	calc_alpha_m = (1.0e+00 / (1.0e+00 + exp((((-6.0e+01) - V_old_) / 5.0e+00))));	//29
	calc_beta_m = ((1.0e-01 / (1.0e+00 + exp(((V_old_ + 3.50e+01) / 5.0e+00)))) + (1.0e-01 / (1.0e+00 + exp(((V_old_ - 5.0e+01) / 2.0e+02)))));	//30
	calc_h_inf = (1.0e+00 / pow((1.0e+00 + exp(((V_old_ + 7.1550e+01) / 7.430e+00))), 2.0e+00));	//33
	calc_alpha_h = (V_old_ < -40) ? 5.70e-02 * exp(((-(V_old_ + 8.0e+01)) / 6.80e+00)) : 0;	//34
	calc_beta_h = (V_old_ < -40) ? (2.70e+00 * exp((7.90e-02 * V_old_))) + (3.10e+05 * exp((3.4850e-01 * V_old_))) : 7.70e-01 / (1.30e-01 * (1.0 + exp(((V_old_ + 1.0660e+01) / (-1.110e+01)))));	//35
	calc_j_inf = (1.0e+00 / pow((1.0e+00 + exp(((V_old_ + 7.1550e+01) / 7.430e+00))), 2.0e+00));	//38
	calc_alpha_j = (V_old_ < -40) ? ((-2.5428e4) * exp(0.2444 * V_old_) - (6.948e-6) * exp(-0.04391 * V_old_)) * (V_old_ + 37.78) / (1. + exp(0.311 * (V_old_ + 79.23))) : 0;	//39
	calc_beta_j = (V_old_ < -40) ? 0.02424 * exp(-0.01052 * V_old_) / (1. + exp(-0.1378 * (V_old_ + 40.14))) : 0.6 * exp((0.057) * V_old_) / (1. + exp(-0.1 * (V_old_ + 32.)));	//40

	calc_d_inf = (1.0e+00 / (1.0e+00 + exp((((-5.0e+00) - V_old_) / 7.50e+00))));	//45
	calc_alpha_d = ((1.40e+00 / (1.0e+00 + exp((((-3.50e+01) - V_old_) / 1.30e+01)))) + 2.50e-01);	//46
	calc_beta_d = (1.40e+00 / (1.0e+00 + exp(((V_old_ + 5.0e+00) / 5.0e+00))));	//47
	calc_gamma_d = (1.0e+00 / (1.0e+00 + exp(((5.0e+01 - V_old_) / 2.0e+01))));	//48

	calc_f_inf = 1. / (1. + exp((V_old_ + 20) / 7));	//51 
	calc_tau_f = 1125 * exp(-(V_old_ + 27) * (V_old_ + 27) / 240) + 80 + 165 / (1. + exp((25 - V_old_) / 10));	//52 300 -> 240 ?
	//calc_tau_f = ( 3.0e+01+3.5e+02/(1.0e+00+exp((V_on_f+2.5e+01)/9.50e+00)) );

	calc_tau_fCa = 2.0;	//58
	calc_s_inf = (1.0e+00 / (1.0e+00 + exp(((V_old_ + 2.0e+01) / 5.0e+00))));	//63
	calc_tau_s = ((8.50e+01 * exp(((-pow((V_old_ + 4.50e+01), 2.0e+00)) / 3.20e+02))) + (5.0e+00 / (1.0e+00 + exp(((V_old_ - 2.0e+01) / 5.0e+00)))) + 3.0e+00);	//64
	calc_r_inf = (1.0e+00 / (1.0e+00 + exp(((2.0e+01 - V_old_) / 6.0e+00))));	//66
	calc_tau_r = ((9.50e+00 * exp(((-pow((V_old_ + 4.0e+01), 2.0e+00)) / 1.80e+03))) + 8.0e-01);	//67
	calc_g_inf = Ca_i_old_ < 3.50e-04 ? 1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.50e-04), 6.0e+00)) : 1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.50e-04), 1.60e+01));	//76
	calc_tau_xr1 = (1.0e+00 * calc_alpha_xr1 * calc_beta_xr1);	//14
	calc_tau_xr2 = (1.0e+00 * calc_alpha_xr2 * calc_beta_xr2);	//19
	calc_tau_xs = (1.0e+00 * calc_alpha_xs * calc_beta_xs);	//25
	calc_tau_m = (1.0e+00 * calc_alpha_m * calc_beta_m);	//31
	calc_tau_h = (1.0e+00 / (calc_alpha_h + calc_beta_h));	//36
	calc_tau_j = (1.0e+00 / (calc_alpha_j + calc_beta_j));	//41
	calc_tau_d = ((1.0e+00 * calc_alpha_d * calc_beta_d) + calc_gamma_d);	//49

	calc_alpha_fCa = (1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.250e-04), 8.0e+00)));	//54
	calc_beta_fCa = (1.0e-01 / (1.0e+00 + exp(((Ca_i_old_ - 5.0e-04) / 1.0e-04))));	//55
	calc_gama_fCa = (2.0e-01 / (1.0e+00 + exp(((Ca_i_old_ - 7.50e-04) / 8.0e-04))));	//56
	calc_fCa_inf = ((calc_alpha_fCa + calc_beta_fCa + calc_gama_fCa + 2.30e-01) / 1.460e+00);	//57

	calc_d_g = ((calc_g_inf - g_old_) / tau_g);	//77
	calc_d_fCa = ((calc_fCa_inf - fCa_old_) / calc_tau_fCa);	//59
}
__device__ void calc_hh_coeff(float* a, float* b, float* pars, float* algs, float* Y_old_, float t)
{
	calc_algs_hh(algs, pars, Y_old_, t);

	Xr1_a_ = -1.0 / calc_tau_xr1;	// 15
	Xr2_a_ = -1.0 / calc_tau_xr2;	// 20
	Xs_a_ = -1.0 / calc_tau_xs;	// 26
	m_a_ = -1.0 / calc_tau_m;	// 32
	h_a_ = -1.0 / calc_tau_h;	// 37
	j_a_ = -1.0 / calc_tau_j;	// 42
	d_a_ = -1.0 / calc_tau_d;	// 50
	f_a_ = -1.0 / calc_tau_f;	// 53
	//fCa_a_= (calc_fCa_inf>fCa_old_&& V_old_>-6.0e+01) ? 0.0 : -1.0/calc_tau_fCa;	// 56
	fCa_a_ = -1.0 / calc_tau_fCa;
	s_a_ = -1.0 / calc_tau_s;	// 64
	r_a_ = -1.0 / calc_tau_r;	// 67
	//g_a_=(calc_g_inf>g_old_&&V_old_>-6.0e+01) ? 0.0 : -1.0/tau_g;
	g_a_ = -1.0 / tau_g;

	Xr1_b_ = (((calc_xr1_inf) / calc_tau_xr1));
	Xr2_b_ = (((calc_xr2_inf) / calc_tau_xr2));
	Xs_b_ = (((calc_xs_inf) / calc_tau_xs));
	m_b_ = (((calc_m_inf) / calc_tau_m));
	h_b_ = (((calc_h_inf) / calc_tau_h));
	j_b_ = (((calc_j_inf) / calc_tau_j));
	d_b_ = (((calc_d_inf) / calc_tau_d));
	f_b_ = (((calc_f_inf) / calc_tau_f));
	//fCa_b_= (calc_fCa_inf>fCa_old_&& V_old_>-6.0e+01) ? 0.0 : calc_fCa_inf/calc_tau_fCa;
	fCa_b_ = calc_fCa_inf / calc_tau_fCa;
	s_b_ = (((calc_s_inf) / calc_tau_s));
	r_b_ = (((calc_r_inf) / calc_tau_r));
	//g_b_ = (calc_g_inf>g_old_&&V_old_>-6.0e+01) ? 0.0 : calc_g_inf/tau_g;
	g_b_ = calc_g_inf / tau_g;

	if ((fCa_old_ * fCa_a_ + fCa_b_) > 0.0 && V_old_ > -37) {
		fCa_a_ = fCa_b_ = 0.0;	// 56
	}
	if ((g_old_ * g_a_ + g_b_) > 0.0 && V_old_ > -37) {
		g_a_ = g_b_ = 0.0;	// 56
	}
}
__device__ void calc_rhs_mk(float* rhs, float* pars, float* algs, float* Y_old_, float t)
{
}
__device__ float calc_stimulus(float* args, float t)
{
	if (stim_state < 0)
		return 0;
	if (stim_state > 0)
		return stim_amplitude;

	float t_since_last_tick = t - floor(t / stim_period) * stim_period;
	float pulse_end = stim_start + stim_duration;
	if (t_since_last_tick >= stim_start && t_since_last_tick <= pulse_end) {
		return stim_amplitude;
	}
	else return 0;
}
__device__ void calc_algs_nl(float* algs, float* args, float* Y_old_, float time,int tid,int N)
{
	
	calc_i_Stim = calc_stimulus(args, time);	//0
	calc_E_Na = (((R * T) / F) * log((Na_o / Na_i_old_)));	//2
	calc_E_K = (((R * T) / F) * log((K_o / K_i_old_)));	//3
	calc_E_Ks = (((R * T) / F) * log(((K_o + (P_kna * Na_o)) / (K_i_old_ + (P_kna * Na_i_old_)))));	//4
	calc_E_Ca = (((5.0e-01 * R * T) / F) * log((Ca_o / Ca_i_old_)));	//5
	calc_i_CaL = ((((g_CaL * d_old_ * f_old_ * fCa_old_ * 4.0e+00 * V_old_ * pow(F, 2.0e+00)) / (R * T)) * ((Ca_i_old_ * exp(((2.0e+00 * V_old_ * F) / (R * T)))) - (3.410e-01 * Ca_o))) / (exp(((2.0e+00 * V_old_ * F) / (R * T))) - 1.0e+00));	//44
	calc_i_NaK = (((((P_NaK * K_o) / (K_o + K_mk)) * Na_i_old_) / (Na_i_old_ + K_mNa)) / (1.0e+00 + (1.2450e-01 * exp((((-1.0e-01) * V_old_ * F) / (R * T)))) + (3.530e-02 * exp((((-V_old_) * F) / (R * T))))));	//69
	calc_i_NaCa = ((K_NaCa * ((exp(((gamma * V_old_ * F) / (R * T))) * pow(Na_i_old_, 3.0e+00) * Ca_o) - (exp((((gamma - 1.0e+00) * V_old_ * F) / (R * T))) * pow(Na_o, 3.0e+00) * Ca_i_old_ * alpha))) / ((pow(Km_Nai, 3.0e+00) + pow(Na_o, 3.0e+00)) * (Km_Ca + Ca_o) * (1.0e+00 + (K_sat * exp((((gamma - 1.0e+00) * V_old_ * F) / (R * T)))))));	//70
	calc_i_p_Ca = ((g_pCa * Ca_i_old_) / (Ca_i_old_ + K_pCa));	//71
	calc_i_rel = ((((a_rel * pow(Ca_SR_old_, 2.0e+00)) / (pow(b_rel, 2.0e+00) + pow(Ca_SR_old_, 2.0e+00))) + c_rel) * d_old_ * g_old_);	//73
	calc_i_up = (Vmax_up / (1.0e+00 + (pow(K_up, 2.0e+00) / pow(Ca_i_old_, 2.0e+00))));	//74
	calc_i_leak = (V_leak * (Ca_SR_old_ - Ca_i_old_));	//75
	calc_Ca_i_bufc = (1.0e+00 / (1.0e+00 + ((Buf_c * K_buf_c) / pow((Ca_i_old_ + K_buf_c), 2.0e+00))));	//79
	calc_Ca_sr_bufsr = (1.0e+00 / (1.0e+00 + ((Buf_sr * K_buf_sr) / pow((Ca_SR_old_ + K_buf_sr), 2.0e+00))));	//80
	calc_i_Kr = (g_Kr * pow((K_o / 5.40e+00), 1.0 / 2.0) * Xr1_old_ * Xr2_old_ * (V_old_ - calc_E_K));	//10
	calc_i_Ks = (g_Ks * pow(Xs_old_, 2.0e+00) * (V_old_ - calc_E_Ks));	//21
	calc_i_Na = (g_Na * pow(m_old_, 3.0e+00) * h_old_ * j_old_ * (V_old_ - calc_E_Na));	//27
	calc_i_b_Na = (g_bna * (V_old_ - calc_E_Na));	//43
	calc_i_b_Ca = (g_bca * (V_old_ - calc_E_Ca));	//61
	calc_i_to = (g_to * r_old_ * s_old_ * (V_old_ - calc_E_K));	//62
	calc_i_p_K = ((g_pK * (V_old_ - calc_E_K)) / (1.0e+00 + exp(((2.50e+01 - V_old_) / 5.980e+00))));	//72
	calc_alpha_K1 = (1.0e-01 / (1.0e+00 + exp((6.0e-02 * ((V_old_ - calc_E_K) - 2.0e+02)))));	//6
	calc_beta_K1 = (((3.0e+00 * exp((2.0e-04 * ((V_old_ - calc_E_K) + 1.0e+02)))) + (1.0e+00 * exp((1.0e-01 * ((V_old_ - calc_E_K) - 1.0e+01))))) / (1.0e+00 + exp(((-5.0e-01) * (V_old_ - calc_E_K)))));	//7
	calc_xK1_inf = (calc_alpha_K1 / (calc_alpha_K1 + calc_beta_K1));	//8
	calc_i_K1 = (g_K1 * calc_xK1_inf * pow((K_o / 5.40e+00), 1.0 / 2.0) * (V_old_ - calc_E_K));	//9
}
__device__ void calc_rhs_nl(float* rhs, float* args, float* algs, float* Y_old_, float t,int tid, int N)
{


	calc_algs_nl(algs, args, Y_old_, t,tid,N);
	float gkatp_f = 4E6 / (1 + pow((atp / 0.25), 2.0)) * (195E-6 / (5E+3)) * pow((K_o / K_i_old_), 0.24);

	//printf("\n%.15f\n", gkatp_f * (V_old_ - 5E3));

	ikatp_f = gkatp_f * (V_old_ - 5E3);
	V_f_ = -(-ikatp_f + calc_i_K1 + +calc_i_Kr + calc_i_Ks + calc_i_CaL + calc_i_NaK + calc_i_Na + calc_i_b_Na + calc_i_NaCa + calc_i_b_Ca + calc_i_p_K + calc_i_p_Ca + calc_i_Stim);
	Ca_i_f_ = ((calc_Ca_i_bufc * (((calc_i_leak - calc_i_up) + calc_i_rel) - (((1.0e+00 * ((calc_i_CaL + calc_i_b_Ca + calc_i_p_Ca) - (2.0e+00 * calc_i_NaCa))) / (2.0e+00 * 1.0e+00 * V_c * F)) * Cm))));	// 81
	Ca_SR_f_ = ((((calc_Ca_sr_bufsr * V_c) / V_sr) * (calc_i_up - (calc_i_rel + calc_i_leak))));	// 82
	Na_i_f_ = ((((-1.0e+00) * (calc_i_Na + calc_i_b_Na + (3.0e+00 * calc_i_NaK) + (3.0e+00 * calc_i_NaCa)) * Cm) / (1.0e+00 * V_c * F)));	// 83
	K_i_f_ = ((((-1.0e+00) * ((calc_i_K1 + calc_i_to + calc_i_Kr + calc_i_Ks + calc_i_p_K + calc_i_Stim) - (2.0e+00 * calc_i_NaK)) * Cm) / (1.0e+00 * V_c * F)));	// 84

}
__device__ void initModel(float* pars, float* Y_old_, float* args)
{

	unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
	unsigned int tid = threadIdx.x + (blockDim.x) * bid;;

	unsigned int N = blockDim.x * gridDim.x * gridDim.y;
	



	printf("\n ss %d %f %f %f %f\n",tid, g_Na,atp,K_o,g_CaL);

	// K_i_old_ = args[0];


	


	V_old_ = -8.620e+01;

	Xr1_old_ = 0.0e+00;

	Xr2_old_ = 1.0e+00;

	Xs_old_ = 0.0e+00;
	m_old_ = 0.0e+00;
	h_old_ = 7.50e-01;
	j_old_ = 7.50e-01;
	d_old_ = 0.0e+00;
	f_old_ = 1.0e+00;
	fCa_old_ = 1.0e+00;
	s_old_ = 1.0e+00;
	r_old_ = 0.0e+00;

	g_old_ = 1.0e+00;
	Ca_i_old_ = 0.00008;//2.0e-04;
	Ca_SR_old_ = 0.56;//2.0e-01;
	Na_i_old_ = 11.6;//1.160e+01;

	K_i_old_ = 138.3;//1.3830e+02;


}
__device__ void step(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut, int tid, int N)
{


	partitionedStep(Y_new_, pars, algs, rhs, Y_old_, t, dt);
	calc_rhs_mk(rhs, pars, algs, Y_old_, t);

	for (int l = MKStart; l < MKEnd; l++)
	{
		Y_new_[l] = Y_old_[l] + dt * rhs[l];


	}

	calc_rhs_nl(rhs, pars, algs, Y_old_, t,tid,N);
	for (int l = NLStart; l < NLEnd; l++)
		Y_new_[l] = Y_old_[l] + dt * rhs[l];
}
__device__ void partitionedStep(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut)
{

	float* as = &(rhs[HHStart]);
	float* bs = &(rhs[nStates]);

	calc_hh_coeff(as, bs, pars, algs, Y_old_, t);

	step(&(Y_new_[HHStart]), nStates_HH, as, bs, &(Y_old_[HHStart]), dt);
}
__device__ void step(float* Y_new_, int n, float* as, float* bs, float* Y_old_, float dt)
{


	for (int i = 0; i < n; i++) {
		if (abs(as[i]) < EPSILON) { // TODO change to epsilon comparison
			Y_new_[i] = Y_old_[i] + dt * (Y_old_[i] * as[i] + bs[i]);
		}
		else {
			float aux = bs[i] / as[i];
			Y_new_[i] = exp(as[i] * dt) * (Y_old_[i] + aux) - aux;
		}
	}

}
__global__ void solveFixed( float* out_g,float dt, float dt_save, float tf, float* args);



//CPU
void  partitionedStepC(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut = NULL);
void stepC(float* Y_new_, int n, float* as, float* bs, float* Y_old_, float dt);
void stepC(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut);
void calc_algs_hhC(float* algs, float* pars, float* Y_old_, float time)
{
	calc_xr1_inf = (1.0e+00 / (1.0e+00 + exp((((-2.60e+01) - V_old_) / 7.0e+00))));	//11+
	calc_alpha_xr1 = (4.50e+02 / (1.0e+00 + exp((((-4.50e+01) - V_old_) / 1.0e+01))));	//12
	calc_beta_xr1 = (6.0e+00 / (1.0e+00 + exp(((V_old_ + 3.0e+01) / 1.150e+01))));	//13
	calc_xr2_inf = (1.0e+00 / (1.0e+00 + exp(((V_old_ + 8.80e+01) / 2.40e+01))));	//16
	calc_alpha_xr2 = (3.0e+00 / (1.0e+00 + exp((((-6.0e+01) - V_old_) / 2.0e+01))));	//17
	calc_beta_xr2 = (1.120e+00 / (1.0e+00 + exp(((V_old_ - 6.0e+01) / 2.0e+01))));	//18
	calc_xs_inf = (1.0e+00 / (1.0e+00 + exp((((-5.0e+00) - V_old_) / 1.40e+01))));	//22
	calc_alpha_xs = (1.10e+03 / pow((1.0e+00 + exp((((-1.0e+01) - V_old_) / 6.0e+00))), 1.0 / 2.0));	//23
	calc_beta_xs = (1.0e+00 / (1.0e+00 + exp(((V_old_ - 6.0e+01) / 2.0e+01))));	//24

	calc_m_inf = (1.0e+00 / pow((1.0e+00 + exp((((-5.6860e+01) - V_old_) / 9.03e+00))), 2.0e+00));	//28
	calc_alpha_m = (1.0e+00 / (1.0e+00 + exp((((-6.0e+01) - V_old_) / 5.0e+00))));	//29
	calc_beta_m = ((1.0e-01 / (1.0e+00 + exp(((V_old_ + 3.50e+01) / 5.0e+00)))) + (1.0e-01 / (1.0e+00 + exp(((V_old_ - 5.0e+01) / 2.0e+02)))));	//30
	calc_h_inf = (1.0e+00 / pow((1.0e+00 + exp(((V_old_ + 7.1550e+01) / 7.430e+00))), 2.0e+00));	//33
	calc_alpha_h = (V_old_ < -40) ? 5.70e-02 * exp(((-(V_old_ + 8.0e+01)) / 6.80e+00)) : 0;	//34
	calc_beta_h = (V_old_ < -40) ? (2.70e+00 * exp((7.90e-02 * V_old_))) + (3.10e+05 * exp((3.4850e-01 * V_old_))) : 7.70e-01 / (1.30e-01 * (1.0 + exp(((V_old_ + 1.0660e+01) / (-1.110e+01)))));	//35
	calc_j_inf = (1.0e+00 / pow((1.0e+00 + exp(((V_old_ + 7.1550e+01) / 7.430e+00))), 2.0e+00));	//38
	calc_alpha_j = (V_old_ < -40) ? ((-2.5428e4) * exp(0.2444 * V_old_) - (6.948e-6) * exp(-0.04391 * V_old_)) * (V_old_ + 37.78) / (1. + exp(0.311 * (V_old_ + 79.23))) : 0;	//39
	calc_beta_j = (V_old_ < -40) ? 0.02424 * exp(-0.01052 * V_old_) / (1. + exp(-0.1378 * (V_old_ + 40.14))) : 0.6 * exp((0.057) * V_old_) / (1. + exp(-0.1 * (V_old_ + 32.)));	//40

	calc_d_inf = (1.0e+00 / (1.0e+00 + exp((((-5.0e+00) - V_old_) / 7.50e+00))));	//45
	calc_alpha_d = ((1.40e+00 / (1.0e+00 + exp((((-3.50e+01) - V_old_) / 1.30e+01)))) + 2.50e-01);	//46
	calc_beta_d = (1.40e+00 / (1.0e+00 + exp(((V_old_ + 5.0e+00) / 5.0e+00))));	//47
	calc_gamma_d = (1.0e+00 / (1.0e+00 + exp(((5.0e+01 - V_old_) / 2.0e+01))));	//48

	calc_f_inf = 1. / (1. + exp((V_old_ + 20) / 7));	//51 
	calc_tau_f = 1125 * exp(-(V_old_ + 27) * (V_old_ + 27) / 240) + 80 + 165 / (1. + exp((25 - V_old_) / 10));	//52 300 -> 240 ?
	//calc_tau_f = ( 3.0e+01+3.5e+02/(1.0e+00+exp((V_on_f+2.5e+01)/9.50e+00)) );

	calc_tau_fCa = 2.0;	//58
	calc_s_inf = (1.0e+00 / (1.0e+00 + exp(((V_old_ + 2.0e+01) / 5.0e+00))));	//63
	calc_tau_s = ((8.50e+01 * exp(((-pow((V_old_ + 4.50e+01), 2.0e+00)) / 3.20e+02))) + (5.0e+00 / (1.0e+00 + exp(((V_old_ - 2.0e+01) / 5.0e+00)))) + 3.0e+00);	//64
	calc_r_inf = (1.0e+00 / (1.0e+00 + exp(((2.0e+01 - V_old_) / 6.0e+00))));	//66
	calc_tau_r = ((9.50e+00 * exp(((-pow((V_old_ + 4.0e+01), 2.0e+00)) / 1.80e+03))) + 8.0e-01);	//67
	calc_g_inf = Ca_i_old_ < 3.50e-04 ? 1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.50e-04), 6.0e+00)) : 1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.50e-04), 1.60e+01));	//76
	calc_tau_xr1 = (1.0e+00 * calc_alpha_xr1 * calc_beta_xr1);	//14
	calc_tau_xr2 = (1.0e+00 * calc_alpha_xr2 * calc_beta_xr2);	//19
	calc_tau_xs = (1.0e+00 * calc_alpha_xs * calc_beta_xs);	//25
	calc_tau_m = (1.0e+00 * calc_alpha_m * calc_beta_m);	//31
	calc_tau_h = (1.0e+00 / (calc_alpha_h + calc_beta_h));	//36
	calc_tau_j = (1.0e+00 / (calc_alpha_j + calc_beta_j));	//41
	calc_tau_d = ((1.0e+00 * calc_alpha_d * calc_beta_d) + calc_gamma_d);	//49

	calc_alpha_fCa = (1.0e+00 / (1.0e+00 + pow((Ca_i_old_ / 3.250e-04), 8.0e+00)));	//54
	calc_beta_fCa = (1.0e-01 / (1.0e+00 + exp(((Ca_i_old_ - 5.0e-04) / 1.0e-04))));	//55
	calc_gama_fCa = (2.0e-01 / (1.0e+00 + exp(((Ca_i_old_ - 7.50e-04) / 8.0e-04))));	//56
	calc_fCa_inf = ((calc_alpha_fCa + calc_beta_fCa + calc_gama_fCa + 2.30e-01) / 1.460e+00);	//57

	calc_d_g = ((calc_g_inf - g_old_) / tau_g);	//77
	calc_d_fCa = ((calc_fCa_inf - fCa_old_) / calc_tau_fCa);	//59
}
void calc_hh_coeffC(float* a, float* b, float* pars, float* algs, float* Y_old_, float t)
{
	calc_algs_hhC(algs, pars, Y_old_, t);

	Xr1_a_ = -1.0 / calc_tau_xr1;	// 15
	Xr2_a_ = -1.0 / calc_tau_xr2;	// 20
	Xs_a_ = -1.0 / calc_tau_xs;	// 26
	m_a_ = -1.0 / calc_tau_m;	// 32
	h_a_ = -1.0 / calc_tau_h;	// 37
	j_a_ = -1.0 / calc_tau_j;	// 42
	d_a_ = -1.0 / calc_tau_d;	// 50
	f_a_ = -1.0 / calc_tau_f;	// 53
	//fCa_a_= (calc_fCa_inf>fCa_old_&& V_old_>-6.0e+01) ? 0.0 : -1.0/calc_tau_fCa;	// 56
	fCa_a_ = -1.0 / calc_tau_fCa;
	s_a_ = -1.0 / calc_tau_s;	// 64
	r_a_ = -1.0 / calc_tau_r;	// 67
	//g_a_=(calc_g_inf>g_old_&&V_old_>-6.0e+01) ? 0.0 : -1.0/tau_g;
	g_a_ = -1.0 / tau_g;

	Xr1_b_ = (((calc_xr1_inf) / calc_tau_xr1));
	Xr2_b_ = (((calc_xr2_inf) / calc_tau_xr2));
	Xs_b_ = (((calc_xs_inf) / calc_tau_xs));
	m_b_ = (((calc_m_inf) / calc_tau_m));
	h_b_ = (((calc_h_inf) / calc_tau_h));
	j_b_ = (((calc_j_inf) / calc_tau_j));
	d_b_ = (((calc_d_inf) / calc_tau_d));
	f_b_ = (((calc_f_inf) / calc_tau_f));
	//fCa_b_= (calc_fCa_inf>fCa_old_&& V_old_>-6.0e+01) ? 0.0 : calc_fCa_inf/calc_tau_fCa;
	fCa_b_ = calc_fCa_inf / calc_tau_fCa;
	s_b_ = (((calc_s_inf) / calc_tau_s));
	r_b_ = (((calc_r_inf) / calc_tau_r));
	//g_b_ = (calc_g_inf>g_old_&&V_old_>-6.0e+01) ? 0.0 : calc_g_inf/tau_g;
	g_b_ = calc_g_inf / tau_g;

	if ((fCa_old_ * fCa_a_ + fCa_b_) > 0.0 && V_old_ > -37) {
		fCa_a_ = fCa_b_ = 0.0;	// 56
	}
	if ((g_old_ * g_a_ + g_b_) > 0.0 && V_old_ > -37) {
		g_a_ = g_b_ = 0.0;	// 56
	}
}
void calc_rhs_mkC(float* rhs, float* pars, float* algs, float* Y_old_, float t)
{
}
float calc_stimulusC(float* pars, float t)
{
	if (stim_state < 0)
		return 0;
	if (stim_state > 0)
		return stim_amplitude;

	float t_since_last_tick = t - floor(t / stim_period) * stim_period;
	float pulse_end = stim_start + stim_duration;
	if (t_since_last_tick >= stim_start && t_since_last_tick <= pulse_end) {
		return stim_amplitude;
	}
	else return 0;
}
void calc_algs_nlC(float* algs, float* args, float* Y_old_, float time,int tid, int N)
{
	calc_i_Stim = calc_stimulusC(args, time);	//0
	calc_E_Na = (((R * T) / F) * log((Na_o / Na_i_old_)));	//2
	calc_E_K = (((R * T) / F) * log((K_o / K_i_old_)));	//3
	calc_E_Ks = (((R * T) / F) * log(((K_o + (P_kna * Na_o)) / (K_i_old_ + (P_kna * Na_i_old_)))));	//4
	calc_E_Ca = (((5.0e-01 * R * T) / F) * log((Ca_o / Ca_i_old_)));	//5
	calc_i_CaL = ((((g_CaL * d_old_ * f_old_ * fCa_old_ * 4.0e+00 * V_old_ * pow(F, 2.0e+00)) / (R * T)) * ((Ca_i_old_ * exp(((2.0e+00 * V_old_ * F) / (R * T)))) - (3.410e-01 * Ca_o))) / (exp(((2.0e+00 * V_old_ * F) / (R * T))) - 1.0e+00));	//44
	calc_i_NaK = (((((P_NaK * K_o) / (K_o + K_mk)) * Na_i_old_) / (Na_i_old_ + K_mNa)) / (1.0e+00 + (1.2450e-01 * exp((((-1.0e-01) * V_old_ * F) / (R * T)))) + (3.530e-02 * exp((((-V_old_) * F) / (R * T))))));	//69
	calc_i_NaCa = ((K_NaCa * ((exp(((gamma * V_old_ * F) / (R * T))) * pow(Na_i_old_, 3.0e+00) * Ca_o) - (exp((((gamma - 1.0e+00) * V_old_ * F) / (R * T))) * pow(Na_o, 3.0e+00) * Ca_i_old_ * alpha))) / ((pow(Km_Nai, 3.0e+00) + pow(Na_o, 3.0e+00)) * (Km_Ca + Ca_o) * (1.0e+00 + (K_sat * exp((((gamma - 1.0e+00) * V_old_ * F) / (R * T)))))));	//70
	calc_i_p_Ca = ((g_pCa * Ca_i_old_) / (Ca_i_old_ + K_pCa));	//71
	calc_i_rel = ((((a_rel * pow(Ca_SR_old_, 2.0e+00)) / (pow(b_rel, 2.0e+00) + pow(Ca_SR_old_, 2.0e+00))) + c_rel) * d_old_ * g_old_);	//73
	calc_i_up = (Vmax_up / (1.0e+00 + (pow(K_up, 2.0e+00) / pow(Ca_i_old_, 2.0e+00))));	//74
	calc_i_leak = (V_leak * (Ca_SR_old_ - Ca_i_old_));	//75
	calc_Ca_i_bufc = (1.0e+00 / (1.0e+00 + ((Buf_c * K_buf_c) / pow((Ca_i_old_ + K_buf_c), 2.0e+00))));	//79
	calc_Ca_sr_bufsr = (1.0e+00 / (1.0e+00 + ((Buf_sr * K_buf_sr) / pow((Ca_SR_old_ + K_buf_sr), 2.0e+00))));	//80
	calc_i_Kr = (g_Kr * pow((K_o / 5.40e+00), 1.0 / 2.0) * Xr1_old_ * Xr2_old_ * (V_old_ - calc_E_K));	//10
	calc_i_Ks = (g_Ks * pow(Xs_old_, 2.0e+00) * (V_old_ - calc_E_Ks));	//21
	calc_i_Na = (g_Na * pow(m_old_, 3.0e+00) * h_old_ * j_old_ * (V_old_ - calc_E_Na));	//27
	calc_i_b_Na = (g_bna * (V_old_ - calc_E_Na));	//43
	calc_i_b_Ca = (g_bca * (V_old_ - calc_E_Ca));	//61
	calc_i_to = (g_to * r_old_ * s_old_ * (V_old_ - calc_E_K));	//62
	calc_i_p_K = ((g_pK * (V_old_ - calc_E_K)) / (1.0e+00 + exp(((2.50e+01 - V_old_) / 5.980e+00))));	//72
	calc_alpha_K1 = (1.0e-01 / (1.0e+00 + exp((6.0e-02 * ((V_old_ - calc_E_K) - 2.0e+02)))));	//6
	calc_beta_K1 = (((3.0e+00 * exp((2.0e-04 * ((V_old_ - calc_E_K) + 1.0e+02)))) + (1.0e+00 * exp((1.0e-01 * ((V_old_ - calc_E_K) - 1.0e+01))))) / (1.0e+00 + exp(((-5.0e-01) * (V_old_ - calc_E_K)))));	//7
	calc_xK1_inf = (calc_alpha_K1 / (calc_alpha_K1 + calc_beta_K1));	//8
	calc_i_K1 = (g_K1 * calc_xK1_inf * pow((K_o / 5.40e+00), 1.0 / 2.0) * (V_old_ - calc_E_K));	//9
}
void calc_rhs_nlC(float* rhs, float* args, float* algs, float* Y_old_, float t,int tid,int  N)
{


	calc_algs_nlC(algs, args, Y_old_, t,tid,N);
	float gkatp_f = 4E6 / (1 + pow((atp / 0.25), 2.0)) * (195E-6 / (5E+3)) * pow((K_o / K_i_old_), 0.24);

	
	ikatp_f = gkatp_f * (V_old_ - 5E3);
	V_f_ = -(-ikatp_f + calc_i_K1 + +calc_i_Kr + calc_i_Ks + calc_i_CaL + calc_i_NaK + calc_i_Na + calc_i_b_Na + calc_i_NaCa + calc_i_b_Ca + calc_i_p_K + calc_i_p_Ca + calc_i_Stim);
	Ca_i_f_ = ((calc_Ca_i_bufc * (((calc_i_leak - calc_i_up) + calc_i_rel) - (((1.0e+00 * ((calc_i_CaL + calc_i_b_Ca + calc_i_p_Ca) - (2.0e+00 * calc_i_NaCa))) / (2.0e+00 * 1.0e+00 * V_c * F)) * Cm))));	// 81
	Ca_SR_f_ = ((((calc_Ca_sr_bufsr * V_c) / V_sr) * (calc_i_up - (calc_i_rel + calc_i_leak))));	// 82
	Na_i_f_ = ((((-1.0e+00) * (calc_i_Na + calc_i_b_Na + (3.0e+00 * calc_i_NaK) + (3.0e+00 * calc_i_NaCa)) * Cm) / (1.0e+00 * V_c * F)));	// 83
	K_i_f_ = ((((-1.0e+00) * ((calc_i_K1 + calc_i_to + calc_i_Kr + calc_i_Ks + calc_i_p_K + calc_i_Stim) - (2.0e+00 * calc_i_NaK)) * Cm) / (1.0e+00 * V_c * F)));	// 84

}
void initModelC(float* pars, float* Y_old_, float* args, int tid, int N)
{




	
	printf("\n ss %d %f %f %f %f\n", tid, g_Na, atp, K_o, g_CaL);

	V_old_ = -8.620e+01;

	Xr1_old_ = 0.0e+00;

	Xr2_old_ = 1.0e+00;

	Xs_old_ = 0.0e+00;
	m_old_ = 0.0e+00;
	h_old_ = 7.50e-01;
	j_old_ = 7.50e-01;
	d_old_ = 0.0e+00;
	f_old_ = 1.0e+00;
	fCa_old_ = 1.0e+00;
	s_old_ = 1.0e+00;
	r_old_ = 0.0e+00;

	g_old_ = 1.0e+00;
	Ca_i_old_ = 0.00008;//2.0e-04;
	Ca_SR_old_ = 0.56;//2.0e-01;
	Na_i_old_ = 11.6;//1.160e+01;

	K_i_old_ = 138.3;//1.3830e+02;


}
void stepC(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut,int tid, int N)
{


	partitionedStepC(Y_new_, pars, algs, rhs, Y_old_, t, dt);
	calc_rhs_mkC(rhs, pars, algs, Y_old_, t);

	for (int l = MKStart; l < MKEnd; l++)
	{
		Y_new_[l] = Y_old_[l] + dt * rhs[l];


	}

	calc_rhs_nlC(rhs, pars, algs, Y_old_, t, tid, N);
	for (int l = NLStart; l < NLEnd; l++)
		Y_new_[l] = Y_old_[l] + dt * rhs[l];
}
void partitionedStepC(float* Y_new_, float* pars, float* algs, float* rhs, float* Y_old_, float t, float dt, float** strut)
{
	float* as = &(rhs[HHStart]);
	float* bs = &(rhs[nStates]);

	calc_hh_coeffC(as, bs, pars, algs, Y_old_, t);

	stepC(&(Y_new_[HHStart]), nStates_HH, as, bs, &(Y_old_[HHStart]), dt);
}
void stepC(float* Y_new_, int n, float* as, float* bs, float* Y_old_, float dt)
{


	for (int i = 0; i < n; i++) {
		if (abs(as[i]) < EPSILON) { // TODO change to epsilon comparison
			Y_new_[i] = Y_old_[i] + dt * (Y_old_[i] * as[i] + bs[i]);
		}
		else {
			float aux = bs[i] / as[i];
			Y_new_[i] = exp(as[i] * dt) * (Y_old_[i] + aux) - aux;
		}
	}

}
void solveFixedCpu(float* out, float dt, float dt_save, float tf, float* args, int tid, int N);


int main(int argc, char** argv)
{

	

	//Simulation Parameters
	OptionParser::addOption("model", "Model: 0 -> ten Tusscher 2004, 1 -> Fox 2002, 2 -> Bondarenko 2004");
	OptionParser::addOption("method", "Method: 0 -> Euler, 1 -> Rush Larsen, 2 -> Euler ADP, 3 -> Rush Larsen ADP, 4 -> UNI");
	OptionParser::addOption("dt", "Base time step.");
	OptionParser::addOption("dt_save", "Time step for saving.");
	OptionParser::addOption("tf", "Final time");
	OptionParser::addOption("ti", "Start output time");
	OptionParser::addOption("dt_max", "Maximum time step for adaptive solvers.");
	OptionParser::addOption("rel_tol", "Relative tolerance for adaptive solvers.");
	OptionParser::addOption("outputFile", "Filename for printing output");
	OptionParser::addOption("n", "Filename for printing output");




	//Parametros 

	OptionParser::addOption("ki", "");
	OptionParser::addOption("ko", "");
	OptionParser::addOption("vmod", "");
	OptionParser::addOption("gna", "");
	OptionParser::addOption("gcl", "");
	OptionParser::addOption("atp", "");
	OptionParser::addOption("use_gpu", "");

	OptionParser::parseOptions(argc, argv);



	float K_o_default = 5.40e+00;
	float g_CaL_default = 1.750e-04;
	float g_Na_default = 1.48380e+01;
	float K_i_default = 138.3; 
	float atp_default = 5.4E0;

	int method_index = OptionParser::foundOption("method") ? OptionParser::parsefloat("method") : 1;
	int model_index = 0;

	float dt = OptionParser::foundOption("dt") ? OptionParser::parsefloat("dt") : 0.1;
	float dt_save = OptionParser::foundOption("dt_save") ? OptionParser::parsefloat("dt_save") : 1;
	float tf = OptionParser::foundOption("tf") ? OptionParser::parsefloat("tf") : 400;
	int N = OptionParser::foundOption("n") ? OptionParser::parseInt("n") : 10000;

	bool use_gpu = OptionParser::foundOption("use_gpu") ? (OptionParser::parseInt("use_gpu")) == 1 ? true : false :  true;

	int np = int(tf / dt_save + 1);

	float* paramS = (float*)malloc(sizeof(float) * 5 * N);
	float* out;
	out = (float*)malloc(sizeof(float) *  np* N);



	ifstream myfile;
	myfile.open("m.txt");


	for (int i = 0; i < N; i++) {
		for (int j = 0; j < 5; j++) {
			myfile >> paramS[i + j * N];

		}
	}


	printf(" \n Problem: %d cells \n",N);
	
	if (use_gpu == true) {
		
		printf(" \n Solve by gpu Grid: %dx%d threads \n", N/10,10);
		float* param_g;
		gpuErrchk(hipMalloc((void**)&param_g, sizeof(float)*4 * 5 * N));
		printf("aa");
		gpuErrchk(hipMemcpy(param_g, paramS, sizeof(float) * 50 * N, hipMemcpyHostToDevice));
		
		int nt = 100;
		int bx=10,by=N/(bx*nt);
		dim3 k(bx,by);
		float* out_g;
		int np = int(tf / dt_save + 1);
		gpuErrchk(hipMalloc((void**)&out_g, sizeof(float) *np  * N));
		solveFixed << < k,nt>> > (out_g, dt, dt_save, tf, param_g);
		gpuErrchk(hipPeekAtLastError());

		gpuErrchk(hipDeviceSynchronize());

		gpuErrchk(hipMemcpy(out, out_g, sizeof(float) * np * N, hipMemcpyDeviceToHost));


		free(out_g);
		free(param_g);
	}

	else {
		printf(" \n Solve by cpu 4 threads \n");
#pragma omp parallel for  num_threads(4)x
		for (int i = 0; i < N; i++)
			solveFixedCpu(out, dt, dt_save, tf, paramS, i, N);

		free(paramS);
	}











	fstream output;
	output.open("out.txt", fstream::out);


	for (int i = 0; i < N; i++) {
		for (int j = 0; j < tf / dt_save; j++) {
			if (j != 0)
				output << " ";
			output << out[j + i * int(tf / dt_save)];
		}
		output << std::endl;
	}



	output.close();


	printf("\n OUTPUT FILE READY!\n");

	free(out); 
	return 0;
}


__global__ void solveFixed(float* out_g,float dt, float dt_save, float tf, float* args)
{
	
	unsigned int bid = blockIdx.y * gridDim.x +blockIdx.x ;
	unsigned int tid = threadIdx.x + (blockDim.x)*bid;
	printf("\n%d %d\n", tid, bid);


	float* Y_old_ = new float[18];
	int np = int(tf / dt_save);

	float* Y_new_ = new float[18];


	float** Tr = NULL;
	if (nStates_MKM_max > 0) {
		Tr = new float* [nStates_MKM_max];
		for (int i = 0; i < nStates_MKM_max; ++i)
			Tr[i] = new float[nStates_MKM_max];
	}
	// rhs will store the righ-hand-side values of NL and MK ODEs, and the coefficients a and b of the HH equations (for the RL method)
	float* rhs = new float[nStates + nStates_HH];
	float* algs = new float[nAlgs];
	
	
	unsigned int N = blockDim.x * gridDim.x * gridDim.y;
	initModel(args, Y_old_, args);

	//	cout << "[ ";
	float aux2 = Y_old_[0];
	float t_save = 0, aux = 0;
	int k = 0;
	for (float t = 0; t <= tf; t += dt) {



		step(Y_new_, args, algs, rhs, Y_old_, t, dt, Tr,tid,N);

		float dv = Y_new_[0] - Y_old_[0];
		if (dv * dv > aux * aux)
			aux = dv;
		for (int l = 0; l < nStates; l++) Y_old_[l] = Y_new_[l];

		t_save += dt;
		if (t_save >= dt_save) {

			out_g[k + tid * np] = Y_new_[0];
			k++;
			t_save = 0;
		//	std:printf("\n%d %d\n",tid, k + tid * np);
			printf("\n%d %f \n",k, Y_new_[0]);
			//	printf("\n%d\n", tid);
			
		}
	
	}
		//out_g[(tid + 1) * np - 1] = aux;

}

void solveFixedCpu(float* out_g, float dt, float dt_save, float tf, float* args, int tid, int N)
{

	float* Y_old_ = new float[18];
	int np = int(tf / dt_save);

	float* Y_new_ = new float[18];


	float** Tr = NULL;
	if (nStates_MKM_max > 0) {
		Tr = new float* [nStates_MKM_max];
		for (int i = 0; i < nStates_MKM_max; ++i)
			Tr[i] = new float[nStates_MKM_max];
	}
	// rhs will store the righ-hand-side values of NL and MK ODEs, and the coefficients a and b of the HH equations (for the RL method)
	float* rhs = new float[nStates + nStates_HH];
	float* algs = new float[nAlgs];
	float* params = new float[48];
	
	initModelC(params, Y_old_, args, tid, N);

	//	cout << "[ ";
	float aux2 = Y_old_[0];
	float t_save = 0, aux = 0;
	int k = 0;
	#pragma unroll
	for (float t = 0; t <= tf; t += dt) {



		stepC(Y_new_,args, algs, rhs, Y_old_, t, dt, Tr,tid,N);

		float dv = Y_new_[0] - Y_old_[0];
		if (dv * dv > aux * aux)
			aux = dv;
		for (int l = 0; l < nStates; l++) Y_old_[l] = Y_new_[l];

		t_save += dt;
		if (t_save >= dt_save) {

			out_g[k + tid * np] = Y_new_[0];
			k++;
			t_save = 0;



			;
		}
		out_g[(tid + 1) * np - 1] = aux;
	}
	//	cout << " -100 " << aux << "]";
}




ArgvParser::ArgvParser()
	: max_key(1),
	help_option(0) // must be smaller than max_key initially

{
	// nothing
}

ArgvParser::~ArgvParser()
{
	// nothing
}

void ArgvParser::reset()
{
	max_key = 1;
	option2key.clear();
	option2attribute.clear();
	option2descr.clear();
	option2value.clear();
	errorcode2descr.clear();
	argument_container.clear();
	intro_description.clear();
	error_option.clear();
	help_option = 0;
}

int ArgvParser::optionKey(const string& _name) const
{
	String2KeyMap::const_iterator it = option2key.find(_name);

	// if not found
	if (it == option2key.end())
		return(-1);

	return(it->second);
}

bool ArgvParser::isDefinedOption(const string& _name) const
{
	return(option2key.find(_name) != option2key.end());
}

bool ArgvParser::foundOption(const string& _name) const
{
	int key = optionKey(_name);

	// not defined -> cannot by found
	if (key == -1)
		return(false);

	// return whether the key of the given option name is in the hash of the
	// parsed options.
	return(option2value.find(key) != option2value.end());
}

string ArgvParser::optionValue(const string& _option) const
{
	int key = optionKey(_option);

	// not defined -> cannot by found
	if (key == -1)
	{
		cerr << "ArgvParser::optionValue(): Requested value of an option the parser did not find or does not know." << endl;
		return("");
	}

	return(option2value.find(key)->second);
}

ArgvParser::ParserResults
ArgvParser::parse(int _argc, char** _argv)
{
	bool finished_options = false; // flag whether an argument was found (options are passed)

	// loop over all command line arguments
	int i = 1; // argument counter
	while (i < _argc)
	{
		string argument = _argv[i];
		unsigned int key = 0;
		string option; // option name
		string value;  // option value

		// if argument is an option
		if (!isValidOptionString(argument))
		{
			// string is a real argument since values are processed elsewhere
			finished_options = true;
			argument_container.push_back(argument);
		}
		else // can be a long or multiple short options at this point
		{
			// check whether we already found an argument
			if (finished_options)
			{
				error_option = argument;
				return(ParserOptionAfterArgument); // return error code
			}
			// check for long options
			if (isValidLongOptionString(argument))
			{
				// handle long options

				// remove trailing '--'
				argument = argument.substr(2);
				// check for option value assignment 'option=value'
				splitOptionAndValue(argument, option, value);

				if (!isDefinedOption(option)) // is this a known option
				{
					error_option = option; // store the option that caused the error
					return(ParserUnknownOption); // return error code if not
				}

				// get the key of this option - now that we know that it is defined
				key = option2key.find(option)->second;
				if (key == help_option) // if help is requested return error code
					return(ParserHelpRequested);

				// do we need to extract a value
				// AND a value is not already assigned from the previous step
				if ((option2attribute.find(key)->second & OptionRequiresValue) && value.empty())
				{
					if (i + 1 >= _argc) // are there arguments left?
					{
						error_option = option; // store the option that caused the error
						return(ParserMissingValue); // the was no argument left although we need a value
					}

					string temp = _argv[i + 1]; // get the next element
					++i; // increase counter now that we moved forward

					if (isValidOptionString(temp))
					{
						error_option = option; // store the option that caused the error
						return(ParserMissingValue);  // missing option value
					}
					value = temp; // assign value
				}
				// add option-value map entry
				option2value[key] = value;
			}
			else // handle short options
			{
				argument = argument.substr(1);   // remove trailing '-'

				// check for option value assignment 'option=value'
				if (splitOptionAndValue(argument, option, value))
				{
					// there was an option <- value assignment
					if (option.length() > 1)
					{
						error_option = option; // store the option that caused the error
						return(ParserMalformedMultipleShortOption); // return error code if option has more than one character
					}

					if (!isDefinedOption(option)) // is this a known option
					{
						error_option = option; // store the option that caused the error
						return(ParserUnknownOption); // return error code if not
					}
					key = option2key.find(option)->second; // get the key for the extracted option name

					if (key == help_option) // if help is requested return error code
						return(ParserHelpRequested);

					// if value is still empty for some reason: we have an error
					if ((option2attribute.find(key)->second & OptionRequiresValue) && value.empty())
					{
						error_option = option; // store the option that caused the error
						return(ParserMissingValue);   // missing option value
					}
					else
						// add option-value map entry
						option2value[key] = value;
				}
				else // no '=' assignment: can be either multiple short options or
					// something like '-s 4'
				{
					// handle short options with value like '-s 4'
					option.clear();
					value.clear();

					if (argument.length() == 1) // if a single short option
					{
						if (!isDefinedOption(argument)) // is this a known option
						{
							error_option = argument; // store the option that caused the error
							return(ParserUnknownOption); // return error code if not
						}
						key = option2key.find(argument)->second; // get the key for the extracted option name

						if (key == help_option) // if help is requested return error code
							return(ParserHelpRequested);

						// check if option needs a value and next arg is not an option
						if ((option2attribute.find(key)->second & OptionRequiresValue))
						{
							if (i + 1 >= _argc) // are there arguments left?
							{
								error_option = argument; // store the option that caused the error
								return(ParserMissingValue); // the was no argument left although we need a value
							}
							string temp = _argv[i + 1]; // get the next element
							++i; // increase counter now that we moved forward

							if (isValidOptionString(temp))
							{
								error_option = argument; // store the option that caused the error
								return(ParserMissingValue);  // missing option value
							}
							// add option-value map entry
							option2value[key] = temp;

						}
						else // no value needed
						{
							option2value[key] = ""; // assign value
						}
					}
					else // handle multiple short option like '-svxgh'
					{
						unsigned int short_option_counter = 0; // position in the multiple short option string
						while (short_option_counter < argument.length()) // parse the whole string
						{
							option = argument[short_option_counter]; // get the option character

							if (!isDefinedOption(option)) // is this a known option
							{
								error_option = option; // store the option that caused the error
								return(ParserUnknownOption); // return error code if not
							}
							key = option2key.find(option)->second; // get the key for the extracted option name

							if (key == help_option) // if help is requested return error code
								return(ParserHelpRequested);

							option2value[key] = value;

							++short_option_counter; // advance one character forward
						}
					}
				}
			}
		}
		++i;
	}

	map<unsigned int, OptionAttributes>::iterator it;
	for (it = option2attribute.begin(); it != option2attribute.end(); it++)
	{
		// if the current option is required look if we got it
		if (it->second & OptionRequired)
		{
			// is the object missing
			if (option2value.find(it->first) == option2value.end())
			{
				// get the list of alternative names for this option
				list<string> alternatives = getAllOptionAlternatives(it->first);

				unsigned int count = 0;
				for (list<string>::const_iterator alt = alternatives.begin();
					alt != alternatives.end();
					++alt)
				{
					++count;
					// additional '-' for long options
					if (alt->length() > 1)
						error_option += "-";

					error_option += "-" + *alt;

					// alternatives to come?
					if (count < alternatives.size())
						error_option += ", "; // add separator
				}
				return(ParserRequiredOptionMissing);
			}
		}
	}

	return(NoParserError); // everthing went fine -> sucess
}

unsigned int ArgvParser::arguments() const
{
	return(argument_container.size());
}

string ArgvParser::argument(unsigned int _id) const
{
	if (_id >= arguments())
	{
		cerr << "ArgvParser::argument(): Request for non-existing argument." << endl;
		return ("");
	}
	else
		return(argument_container[_id]);
}

const vector<string>& ArgvParser::allArguments() const
{
	return(argument_container);
}

const string& ArgvParser::errorOption() const
{
	return(error_option);
}

std::string ArgvParser::parseErrorDescription(ParserResults _error_code) const
{
	string descr;

	switch (_error_code)
	{
	case ArgvParser::NoParserError:
		// no error -> nothing to do
		break;
	case ArgvParser::ParserUnknownOption:
		descr = "Unknown option: '" + errorOption() + "'";
		break;
	case ArgvParser::ParserMissingValue:
		descr = "Missing required value for option: '" + errorOption() + "'";
		break;
	case ArgvParser::ParserOptionAfterArgument:
		descr = "Misplaced option '" + errorOption() + "' detected. All option have to be BEFORE the first argument";
		break;
	case ArgvParser::ParserMalformedMultipleShortOption:
		descr = "Malformed short-options: '" + errorOption() + "'";
		break;
	case ArgvParser::ArgvParser::ParserRequiredOptionMissing:
		descr = "Required option missing: '" + errorOption() + "'";
		break;
	case ArgvParser::ParserHelpRequested: // help

		break;
	default:
		cerr << "ArgvParser::documentParserErrors(): Unknown error code" << endl;
	}

	return(descr);
}

bool ArgvParser::defineOption(const string& _name,
	const string& _descr,
	OptionAttributes _attrs)
{
	// do nothing if there already is an option of this name
	if (isDefinedOption(_name))
	{
		cerr << "ArgvParser::defineOption(): The option label equals an already defined option." << endl;
		return(false);
	}

	// no digits as short options allowed
	if (_name.length() == 1 && isDigit(_name[0]))
	{
		cerr << "ArgvParser::defineOption(): Digits as short option labels are not allowd." << endl;
		return(false);
	}

	option2key[_name] = max_key;     // give the option a unique key

	// store the option attributes
	option2attribute[max_key] = _attrs;

	// store the option description if there is one
	if (_descr.length())
		option2descr[max_key] = _descr;

	// inc the key counter
	++max_key;

	return(true);
}

bool ArgvParser::defineOptionAlternative(const string& _original,
	const string& _alternative)
{
	// do nothing if there already is no option of this name
	if (!isDefinedOption(_original))
	{
		cerr << "ArgvParser::defineOptionAlternative(): Original option label is not a defined option." << endl;
		return(false);
	}

	// AND no digits as short options allowed
	if (_alternative.length() == 1 && isDigit(_alternative[0]))
	{
		cerr << "ArgvParser::defineOptionAlternative(): Digits as short option labels are not allowd." << endl;
		return(false);
	}

	// AND do nothing if there already is an option with the alternativ name
	if (isDefinedOption(_alternative))
	{
		cerr << "ArgvParser::defineOptionAlternative(): The alternative option label equals an already defined option." << endl;
		return(false);
	}

	option2key[_alternative] = optionKey(_original);

	return(true);
}


bool ArgvParser::setHelpOption(const string& _shortname,
	const string& _longname,
	const string& _descr)
{
	// do nothing if any name is already in use
	if (isDefinedOption(_shortname) || isDefinedOption(_longname))
	{
		cerr << "ArgvParser::setHelpOption(): Short or long help option label equals an already defined option." << endl;
		return(false);
	}

	// define the help option's short name and the alternative
	// longname
	defineOption(_shortname, _descr, NoOptionAttribute);
	defineOptionAlternative(_shortname, _longname);

	help_option = max_key - 1; // store the key in a special member

	return(true);
}

void ArgvParser::addErrorCode(int _code, const string& _descr)
{
	errorcode2descr[_code] = _descr;
}

void ArgvParser::setIntroductoryDescription(const string& _descr)
{
	intro_description = _descr;
}

list<string> ArgvParser::getAllOptionAlternatives(unsigned int _key) const
{
	// keys go here
	list<string> keys;
	// for all container elements
	for (map<string, unsigned int>::const_iterator it = option2key.begin();
		it != option2key.end();
		++it)
	{
		if (it->second == _key)
			keys.push_back(it->first);
	}
	return(keys);
}

bool CommandLineProcessing::isDigit(const char& _char)
{
	if (_char == '0' || _char == '1' || _char == '2' || _char == '3'
		|| _char == '4' || _char == '5' || _char == '6' || _char == '7'
		|| _char == '8' || _char == '9')
		return(true);
	else
		return(false);
}

bool CommandLineProcessing::isValidOptionString(const string& _string)
{
	// minimal short option length is 2
	if (_string.length() < 2)
		return(false);

	// is it an option (check for '-' as first character)
	if (_string.compare(0, 1, "-"))
		return(false);

	// not an option if just '--'
	if (_string.length() == 2 && _string == "--")
		return(false);

	// it might still be a negative number
	// (but not if there is no digit afterwards)
	if (isDigit(_string[1]))
		return(false);

	// let's consider this an option
	return(true);
}

bool CommandLineProcessing::isValidLongOptionString(const string& _string)
{
	if (_string.length() < 4) // must be at least '--??'
		return(false);

	// is it an option (check for '--')
	if (_string.compare(0, 2, "--"))
		return(false);
	else
		return(true);
}

bool CommandLineProcessing::splitOptionAndValue(const string& _string,
	string& _option, string& _value)
{
	// string token container
	std::vector<string> tokens;

	// split string by '=' delimiter
	splitString(tokens, _string, "=");

	// check for option value assignment 'option=value'
	if (tokens.size() < 2)
	{
		_option = _string; // the option is the whole string
		return(false);
	}

	// separate option and value
	_option = tokens[0];

	// concat all remaining tokens to the value string
	for (unsigned int i = 1; i < tokens.size(); ++i)
	{
		_value.append(tokens[i]);
	}

	return(true);
}

string CommandLineProcessing::trimmedString(const std::string& _str)
{
	// no string no work
	if (_str.length() == 0)
		return _str;

	string::size_type start_pos = _str.find_first_not_of(" \a\b\f\n\r\t\v");
	string::size_type end_pos = _str.find_last_not_of(" \a\b\f\n\r\t\v");

	// check whether there was any non-whitespace
	if (start_pos == string::npos)
		return("");

	return string(_str, start_pos, end_pos - start_pos + 1);
}

bool CommandLineProcessing::expandRangeStringToUInt(const std::string& _string,
	std::vector< unsigned int >& _expanded)
{
	list<string> tokens;
	// split string by delimiter
	splitString(tokens, _string, ",");

	// loop over all entries
	for (list<string>::const_iterator it = tokens.begin(); it != tokens.end(); it++)
	{
		const string& entry = *it; // convenience reference

#ifdef ARGVPARSER_DEBUG

		cout << "TOKEN: " << entry << endl;
#endif

		// if range was given
		if (entry.find("-") != string::npos)
		{
			// split into upper and lower border
			list<string> range_borders;
			splitString(range_borders, entry, "-");

			// fail if insane range spec
			if (range_borders.size() != 2)
				return(false);

			int first = atoi(range_borders.begin()->c_str());
			int second = atoi((++range_borders.begin())->c_str());

			// write id in increasing order
			if (first <= second)

			{
				for (int j = first; j <= second; ++j)
				{
					_expanded.push_back(j);
				}
			}
			else // write id in decreasing order
			{
				for (int k = first; k >= second; k--)
				{
					_expanded.push_back(k);
				}
			}
		}
		else // single number was given
			_expanded.push_back(atoi(entry.c_str())); // store id
	}

	return(true);
}

std::string CommandLineProcessing::formatString(const std::string& _string,
	unsigned int _width,
	unsigned int _indent)
{
	// if insane parameters do nothing
	if (_indent >= _width)
		return(_string);

	// list of lines of the formated string
	list<string> lines;

	// current position in the string
	unsigned int pos = 0;

	// till the end of the string
	while (pos < _string.length())
	{
		// get the next line of the string
		string line = _string.substr(pos, _width - _indent);

#ifdef ARGVPARSER_DEBUG

		cout << "EXTRACT: '" << line << "'" << endl;
#endif

		// check for newlines in the line and break line at first occurence (if any)
		string::size_type first_newline = line.find_first_of("\n");
		if (first_newline != string::npos)
		{
			line = line.substr(0, first_newline);
		}

		// we need to check for possible breaks within words only if the extracted
		// line spans the whole allowed width
		bool check_truncation = true;
		if (line.length() < _width - _indent)
			check_truncation = false;

		// remove unecessary whitespace at front and back
		line = trimmedString(line);

#ifdef ARGVPARSER_DEBUG

		cout << "TRIMMED: '" << line << "'" << endl;
#endif

		// only perform truncation if there was enough data for a full line
		if (!check_truncation)
			pos += line.length() + 1;
		else
		{
			// look for the last whitespace character
			string::size_type last_white_space = line.find_last_of(" \a\b\f\n\r\t\v");

			if (last_white_space != string::npos) // whitespace found!
			{
				// truncated the line at the last whitespace
				line = string(line, 0, last_white_space);
				pos += last_white_space + 1;
			}
			else // no whitespace found
				// rude break! we can leave the line in its current state
				pos += _width - _indent;
		}

		if (!line.empty())
		{
#ifdef ARGVPARSER_DEBUG
			cout << "UNINDEN: '" << line << "'" << endl;
#endif

			if (_indent)
				line.insert(0, _indent, ' ');

#ifdef ARGVPARSER_DEBUG

			cout << "INDENT: '" << line << "'" << endl;
#endif

			lines.push_back(line);
		}
	}

	// concat the formated string
	string formated;
	bool first = true;
	// for all lines
	for (list<string>::iterator it = lines.begin(); it != lines.end(); ++it)
	{
		// prefix with newline if not first
		if (!first)
			formated += "\n";
		else
			first = false;

		formated += *it;
	}
	return(formated);
}

ArgvParser OptionParser::cmd;

void OptionParser::addOption(const string& option, const string& desc) {
	cmd.defineOption(option, desc, ArgvParser::NoOptionAttribute);
}

void OptionParser::setup() {
	//define error codes
	cmd.addErrorCode(0, "Success");
	cmd.addErrorCode(1, "Error");
}

void OptionParser::parseOptions(int argc, char** argv) {
	int result = cmd.parse(argc, argv);

	if (result != ArgvParser::NoParserError) {
		cout << "Parse ERROR!!!: " << cmd.parseErrorDescription(result) << endl;
		exit(1);
	}
}

string OptionParser::optionValue(const string& option) {
	return cmd.optionValue(option);
}

bool OptionParser::foundOption(const string& option) {
	return cmd.foundOption(option);
}

OptionParser::OptionParser() {
	this->myCmd.addErrorCode(0, "Success");
	this->myCmd.addErrorCode(1, "Error");
}

void OptionParser::add(const string& option, const string& desc) {
	myCmd.defineOption(option, desc, ArgvParser::NoOptionAttribute);
}

int OptionParser::parse(string optionsText) {
	vector<char*> args;
	istringstream iss(optionsText);
	string token;
	int optionsCount = 0;
	while (iss >> token) {
		char* arg = new char[token.size() + 1];
		copy(token.begin(), token.end(), arg);
		arg[token.size()] = '\0';
		args.push_back(arg);
		optionsCount++;
	}
	args.push_back(0);
	int result = myCmd.parse(optionsCount, &args[0]);

	if (result != ArgvParser::NoParserError) {
		cout << "Parse ERROR!!!: " << myCmd.parseErrorDescription(result) << endl;
		return 0;
	}
	return 1;
}
string OptionParser::value(const string& option) {
	return myCmd.optionValue(option);
}
bool OptionParser::has(const string& option) {
	return myCmd.foundOption(option);
}

float OptionParser::parsefloat(const string& option) {
	float aux;
	sscanf(cmd.optionValue(option).c_str(), "%lf", &aux);
	return aux;
}

int OptionParser::parseInt(const string& option) {
	int aux;
	sscanf(cmd.optionValue(option).c_str(), "%d", &aux);
	return aux;
}
